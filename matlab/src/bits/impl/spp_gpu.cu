#include "hip/hip_runtime.h"
// @file spp_gpu.cu
// @brief  SPP block implementation (GPU)
// @author Hakan Bilen

#include "spp.hpp"
#include "../datacu.hpp"
#include <assert.h>
#include <float.h>
#include <sm_20_atomic_functions.h>
/* ---------------------------------------------------------------- */
/*                                              spp_average_forward */
/* ---------------------------------------------------------------- */
template<typename T> __global__ void
spp_average_kernel
(T* pooled,
 const T* data,
 const int height,
 const int width,
 const int depth,
 const int size,
 const int numTotBins,
 const int numLevels,
 const T* levels,
 const int numROIs,
 const T* ROIs)
{
  int pooledIndex = threadIdx.x + blockIdx.x * blockDim.x;
  int pooledVolume = numTotBins * depth * numROIs;

  if (pooledIndex < pooledVolume) {

    int pl = pooledIndex % numTotBins;
    int pc = (pooledIndex / numTotBins) % depth;
    int pr = (pooledIndex / numTotBins / depth); // roi no

    int roi_image   = ROIs[5 * pr + 0];

    int roi_start_h = ROIs[5 * pr + 1];
    int roi_start_w = ROIs[5 * pr + 2];
    int roi_end_h   = ROIs[5 * pr + 3];
    int roi_end_w   = ROIs[5 * pr + 4];

    if(roi_start_w==roi_end_w) {
      if(roi_start_w>0)
        roi_start_w--;
      else
        roi_end_w++;
    }
    if(roi_start_h==roi_end_h) {
      if(roi_start_h>0)
        roi_start_h--;
      else
        roi_end_h++;
    }

    // Force malformed ROIs to be 1x1
    int roi_width = max(roi_end_w - roi_start_w + 1, 1);
    int roi_height = max(roi_end_h - roi_start_h + 1, 1);

    // Find pyramid level and bin
    int pb = -1;
    int pLevel = -1;
    int numBins = 0;
    for(int l=0;l<numLevels;l++) {
      if(pl-numBins>=0 && pl-numBins<static_cast<int>(levels[l] * levels[l])) {
        pb = pl - numBins;
        pLevel = l;
      }
      numBins += static_cast<int>(levels[l] * levels[l]);
    }

    int pooledWidth  = levels[pLevel];
    int pooledHeight = levels[pLevel];
    int pw = pb / pooledHeight;
    int ph = pb % pooledHeight;


    const T bin_size_h = static_cast<T>(roi_height)
        / static_cast<T>(pooledHeight);
    const T bin_size_w = static_cast<T>(roi_width)
        / static_cast<T>(pooledWidth);


    int hstart = static_cast<int>(floor(static_cast<T>(ph)
                                        * bin_size_h));
    int wstart = static_cast<int>(floor(static_cast<T>(pw)
                                        * bin_size_w));
    int hend = static_cast<int>(ceil(static_cast<T>(ph + 1)
                                     * bin_size_h));
    int wend = static_cast<int>(ceil(static_cast<T>(pw + 1)
                                     * bin_size_w));

    // Add roi offsets and clip to input boundaries
    hstart = min(max(hstart + roi_start_h, 0), height);
    hend = min(max(hend + roi_start_h, 0), height);
    wstart = min(max(wstart + roi_start_w, 0), width);
    wend = min(max(wend + roi_start_w, 0), width);

    int offset_data = (roi_image * depth + pc) * (width*height);

    data += offset_data;
    T bestValue = 0;
    const T coef = 1.f / (T)((wend-wstart) * (hend-hstart));
    for (int w = wstart; w < wend; ++w) {
      for (int h = hstart; h < hend; ++h) {
        int index = w * height + h ;
        bestValue += data[index] * coef;
      }
    }
    pooled[pooledIndex] = bestValue ;
  }
}

/* ---------------------------------------------------------------- */
/*                                                  spp_max_forward */
/* ---------------------------------------------------------------- */

template<typename T> __global__ void
spp_max_kernel(T* pooled,
 const T* data,
 const int height,
 const int width,
 const int depth,
 const int size,
 const int numTotBins,
 const int numLevels,
 const T* levels,
 const int numROIs,
 const T* ROIs)
{
  int pooledIndex = threadIdx.x + blockIdx.x * blockDim.x;


  int pooledVolume = numTotBins * depth * numROIs;

  if (pooledIndex < pooledVolume) {

    int pl = pooledIndex % numTotBins;
    int pc = (pooledIndex / numTotBins) % depth;
    int pr = (pooledIndex / numTotBins / depth); // roi no
    int roi_image   = ROIs[5 * pr + 0];


    int roi_start_h = ROIs[5 * pr + 1];
    int roi_start_w = ROIs[5 * pr + 2];
    int roi_end_h   = ROIs[5 * pr + 3];
    int roi_end_w   = ROIs[5 * pr + 4];

    if(roi_start_w==roi_end_w) {
      if(roi_start_w>0)
        roi_start_w--;
      else
        roi_end_w++;
    }
    if(roi_start_h==roi_end_h) {
      if(roi_start_h>0)
        roi_start_h--;
      else
        roi_end_h++;
    }


    // Force malformed ROIs to be 1x1
    int roi_width = max(roi_end_w - roi_start_w + 1, 1);
    int roi_height = max(roi_end_h - roi_start_h + 1, 1);

    // Find pyramid level and bin
    int pb = -1;
    int pLevel = -1;
    int numBins = 0;
    for(int l=0;l<numLevels;l++) {
      if(pl-numBins>=0 && pl-numBins<static_cast<int>(levels[l] * levels[l])) {
        pb = pl - numBins;
        pLevel = l;
      }
      numBins += static_cast<int>(levels[l] * levels[l]);
    }

    int pooledWidth  = levels[pLevel];
    int pooledHeight = levels[pLevel];
    int pw = pb / pooledHeight;
    int ph = pb % pooledHeight;


    const T bin_size_h = static_cast<T>(roi_height)
        / static_cast<T>(pooledHeight);
    const T bin_size_w = static_cast<T>(roi_width)
        / static_cast<T>(pooledWidth);


    int hstart = static_cast<int>(floor(static_cast<T>(ph)
                                        * bin_size_h));
    int wstart = static_cast<int>(floor(static_cast<T>(pw)
                                        * bin_size_w));
    int hend = static_cast<int>(ceil(static_cast<T>(ph + 1)
                                     * bin_size_h));
    int wend = static_cast<int>(ceil(static_cast<T>(pw + 1)
                                     * bin_size_w));

    // Add roi offsets and clip to input boundaries
    hstart = min(max(hstart + roi_start_h, 0), height);
    hend = min(max(hend + roi_start_h, 0), height);
    wstart = min(max(wstart + roi_start_w, 0), width);
    wend = min(max(wend + roi_start_w, 0), width);
    bool is_empty = (hend <= hstart) || (wend <= wstart);

    int offset_data = (roi_image * depth + pc) * (width*height);

    data += offset_data;
    T bestValue = is_empty ? 0 : data[wstart * height + hstart];
    for (int w = wstart; w < wend; ++w) {
      for (int h = hstart; h < hend; ++h) {
        int index = w * height + h ;
        bestValue = max(bestValue, data[index]) ;
      }
    }
    pooled[pooledIndex] = bestValue ;

  }
}


/* ---------------------------------------------------------------- */
/*                                                 spp_max_backward */
/* ---------------------------------------------------------------- */

// an implementation of atomicAdd() for double (really slow)
static __device__ double atomicAdd(double* address, double val)
{
  unsigned long long int* address_as_ull = (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;
  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed,
                    __double_as_longlong(val +
                                         __longlong_as_double(assumed)));
  } while (assumed != old);
  return __longlong_as_double(old);
}

template<typename T> __global__ void
spp_max_backward_kernel(T* derData,
                        const T* data,
                        const T* derPooled,
                        const int height,
                        const int width,
                        const int depth,
                        const int size,
                        const int numTotBins,
                        const int numLevels,
                        const T* levels,
                        const int numROIs,
                        const T* ROIs)
{
  int pooledIndex = threadIdx.x + blockIdx.x * blockDim.x;

  int pooledVolume = numTotBins * depth * numROIs;

  if (pooledIndex < pooledVolume) {


    int pl = pooledIndex % numTotBins;
    int pc = (pooledIndex / numTotBins) % depth;
    int pr = (pooledIndex / numTotBins / depth); // roi no

    int roi_image   = ROIs[5 * pr + 0];

    int roi_start_h = ROIs[5 * pr + 1];
    int roi_start_w = ROIs[5 * pr + 2];
    int roi_end_h   = ROIs[5 * pr + 3];
    int roi_end_w   = ROIs[5 * pr + 4];

    if(roi_start_w==roi_end_w) {
      if(roi_start_w>0)
        roi_start_w--;
      else
        roi_end_w++;
    }
    if(roi_start_h==roi_end_h) {
      if(roi_start_h>0)
        roi_start_h--;
      else
        roi_end_h++;
    }

    // Force malformed ROIs to be 1x1
    int roi_width = max(roi_end_w - roi_start_w + 1, 1);
    int roi_height = max(roi_end_h - roi_start_h + 1, 1);

    // Find pyramid level and bin
    int pb = -1;
    int pLevel = -1;
    int numBins = 0;
    for(int l=0;l<numLevels;l++) {
      if(pl-numBins>=0 && pl-numBins<static_cast<int>(levels[l] * levels[l])) {
        pb = pl - numBins;
        pLevel = l;
      }
      numBins += static_cast<int>(levels[l] * levels[l]);
    }

    int pooledWidth  = levels[pLevel];
    int pooledHeight = levels[pLevel];
    int pw = pb / pooledHeight;
    int ph = pb % pooledHeight;


    const T bin_size_h = static_cast<T>(roi_height)
        / static_cast<T>(pooledHeight);
    const T bin_size_w = static_cast<T>(roi_width)
        / static_cast<T>(pooledWidth);



    int hstart = static_cast<int>(floor(static_cast<T>(ph)
                                        * bin_size_h));
    int wstart = static_cast<int>(floor(static_cast<T>(pw)
                                        * bin_size_w));
    int hend = static_cast<int>(ceil(static_cast<T>(ph + 1)
                                     * bin_size_h));
    int wend = static_cast<int>(ceil(static_cast<T>(pw + 1)
                                     * bin_size_w));

    // Add roi offsets and clip to input boundaries
    hstart = min(max(hstart + roi_start_h, 0), height);
    hend = min(max(hend + roi_start_h, 0), height);
    wstart = min(max(wstart + roi_start_w, 0), width);
    wend = min(max(wend + roi_start_w, 0), width);
    bool is_empty = (hend <= hstart) || (wend <= wstart);

    data += (roi_image * depth + pc) * (width*height);
    derData += (roi_image * depth + pc) * (width*height);

    int bestIndex = wstart * height + hstart;
    T bestValue = is_empty ? 0 : data[bestIndex];
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        int index = w * height + h ;
        T value = data[index] ;
        if (value > bestValue) {
          bestValue = value ;
          bestIndex = index ;
        }
      }
    }

    /*
     This is bad, but required to eliminate a race condition when writing
     to bottom_diff.
     Caffe goes the other way around, but requires remembering the layer
     output, or the maximal indexes.
     atomicAdd(add, val)
     */
    atomicAdd(derData + bestIndex, derPooled[pooledIndex]) ;
  }
}


/* ---------------------------------------------------------------- */
/*                                             spp_average_backward */
/* ---------------------------------------------------------------- */
template<typename T> __global__ void
spp_average_backward_kernel
(T* derData,
 const T* data,
 const T* derPooled,
 const int height,
 const int width,
 const int depth,
 const int size,
 const int numTotBins,
 const int numLevels,
 const T* levels,
 const int numROIs,
 const T* ROIs)
{
  int pooledIndex = threadIdx.x + blockIdx.x * blockDim.x;


  int pooledVolume = numTotBins * depth * numROIs;

  if (pooledIndex < pooledVolume) {

    int pl = pooledIndex % numTotBins;
    int pc = (pooledIndex / numTotBins) % depth;
    int pr = (pooledIndex / numTotBins / depth); // roi no

    int roi_image   = ROIs[5 * pr + 0];

    int roi_start_h = ROIs[5 * pr + 1];
    int roi_start_w = ROIs[5 * pr + 2];
    int roi_end_h   = ROIs[5 * pr + 3];
    int roi_end_w   = ROIs[5 * pr + 4];

    if(roi_start_w==roi_end_w) {
      if(roi_start_w>0)
        roi_start_w--;
      else
        roi_end_w++;
    }
    if(roi_start_h==roi_end_h) {
      if(roi_start_h>0)
        roi_start_h--;
      else
        roi_end_h++;
    }

    // Force malformed ROIs to be 1x1
    int roi_width = max(roi_end_w - roi_start_w + 1, 1);
    int roi_height = max(roi_end_h - roi_start_h + 1, 1);

    // Find pyramid level and bin
    int pb = -1;
    int pLevel = -1;
    int numBins = 0;
    for(int l=0;l<numLevels;l++) {
      if(pl-numBins>=0 && pl-numBins<static_cast<int>(levels[l] * levels[l])) {
        pb = pl - numBins;
        pLevel = l;
      }
      numBins += static_cast<int>(levels[l] * levels[l]);
    }
    int pooledWidth  = levels[pLevel];
    int pooledHeight = levels[pLevel];
    int pw = pb / pooledHeight;
    int ph = pb % pooledHeight;


    const T bin_size_h = static_cast<T>(roi_height)
        / static_cast<T>(pooledHeight);
    const T bin_size_w = static_cast<T>(roi_width)
        / static_cast<T>(pooledWidth);


    int hstart = static_cast<int>(floor(static_cast<T>(ph)
                                        * bin_size_h));
    int wstart = static_cast<int>(floor(static_cast<T>(pw)
                                        * bin_size_w));
    int hend = static_cast<int>(ceil(static_cast<T>(ph + 1)
                                     * bin_size_h));
    int wend = static_cast<int>(ceil(static_cast<T>(pw + 1)
                                     * bin_size_w));

    // Add roi offsets and clip to input boundaries
    hstart = min(max(hstart + roi_start_h, 0), height);
    hend = min(max(hend + roi_start_h, 0), height);
    wstart = min(max(wstart + roi_start_w, 0), width);
    wend = min(max(wend + roi_start_w, 0), width);

    data += (roi_image * depth + pc) * (width*height);
    derData += (roi_image * depth + pc) * (width*height);

    const T coef = 1.f / (T)((wend-wstart)*(hend-hstart));
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        int index = w * height + h ;
      /*
       This is bad, but required to eliminate a race condition when writing
       to bottom_diff.
       Caffe goes the other way around, but requires remembering the layer
       output, or the maximal indexes.
       atomicAdd(add, val)
       */
        atomicAdd(derData + index, derPooled[pooledIndex] * coef) ;
      }
    }


  }
}
/* ---------------------------------------------------------------- */
/*                                                        Interface */
/* ---------------------------------------------------------------- */
namespace vl { namespace impl {

  template <typename type>
  struct spp_max<vl::VLDT_GPU, type>
  {
    static vl::ErrorCode
    forward(type* pooled,
            type const* data,
            size_t height, size_t width, size_t depth, size_t size,
            size_t numTotBins,
            size_t numLevels, type const* levels,
            size_t numROIs, type const* ROIs)
  {
    int pooledVolume = numTotBins * depth * numROIs;

    spp_max_kernel<type>
      <<< divideAndRoundUp(pooledVolume, VL_CUDA_NUM_THREADS),VL_CUDA_NUM_THREADS >>>
      (pooled, data,
       height, width, depth, size,
       numTotBins,
       numLevels, levels,
       numROIs, ROIs);

      hipError_t status = hipPeekAtLastError() ;
      return (status == hipSuccess) ? vl::VLE_Success : vl::VLE_Cuda ;
    }

    static vl::ErrorCode
    backward(type* derData,
             type const* data,
             type const* derPooled,
             size_t height, size_t width, size_t depth, size_t size,
             size_t numTotBins,
             size_t numLevels, type const* levels,
             size_t numROIs, type const* ROIs)
    {
      int pooledVolume = numTotBins * depth * numROIs;

      spp_max_backward_kernel<type>
          <<< divideAndRoundUp(pooledVolume, VL_CUDA_NUM_THREADS), VL_CUDA_NUM_THREADS >>>
          (derData, data, derPooled,
           height, width, depth, size,
           numTotBins,
           numLevels, levels,
           numROIs, ROIs);

      hipError_t status = hipPeekAtLastError() ;
      return (status == hipSuccess) ? vl::VLE_Success : vl::VLE_Cuda ;
    }
  } ; // spp_max

  template <typename type>
  struct spp_average<vl::VLDT_GPU, type>
  {
    static vl::ErrorCode
    forward(type* pooled,
            type const* data,
            size_t height, size_t width, size_t depth, size_t size,
            size_t numTotBins,
            size_t numLevels, type const* levels,
            size_t numROIs, type const* ROIs)
  {
    int pooledVolume = numTotBins * depth * numROIs;

    spp_average_kernel<type>
      <<< divideAndRoundUp(pooledVolume, VL_CUDA_NUM_THREADS),VL_CUDA_NUM_THREADS >>>
      (pooled, data,
       height, width, depth, size,
       numTotBins,
       numLevels, levels,
       numROIs, ROIs);

      hipError_t status = hipPeekAtLastError() ;
      return (status == hipSuccess) ? vl::VLE_Success : vl::VLE_Cuda ;
    }

    static vl::ErrorCode
    backward(type* derData,
             type const* data,
             type const* derPooled,
             size_t height, size_t width, size_t depth, size_t size,
             size_t numTotBins,
             size_t numLevels, type const * levels,
             size_t numROIs, type const * ROIs)
    {
      int pooledVolume = numTotBins * depth * numROIs;

      spp_average_backward_kernel<type>
          <<< divideAndRoundUp(pooledVolume, VL_CUDA_NUM_THREADS), VL_CUDA_NUM_THREADS >>>
          (derData, data, derPooled,
           height, width, depth, size,
           numTotBins,
           numLevels, levels,
           numROIs, ROIs);

      hipError_t status = hipPeekAtLastError() ;
      return (status == hipSuccess) ? vl::VLE_Success : vl::VLE_Cuda ;
    }
  } ; // spp_average
} } ; // namespace vl::impl

// Instantiations
template struct vl::impl::spp_max<vl::VLDT_GPU, float> ;
template struct vl::impl::spp_average<vl::VLDT_GPU, float> ;

#ifdef ENABLE_DOUBLE
template struct vl::impl::spp_max<vl::VLDT_GPU, double> ;
template struct vl::impl::spp_average<vl::VLDT_GPU, double> ;
#endif
