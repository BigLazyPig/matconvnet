#include "hip/hip_runtime.h"
// @file nnspp.cu
// @brief SPP block
// @author Hakan Bilen 

/*
Copyright (C) 2014-16 Andrea Vedaldi and Karel Lenc.
All rights reserved.

This file is part of the VLFeat library and is made available under
the terms of the BSD license (see the COPYING file).
*/


#include "nnspp.hpp"
#include "impl/spp.hpp"

#if ENABLE_GPU
#include "datacu.hpp"
#endif

#include <assert.h>

using namespace vl ;

/* ---------------------------------------------------------------- */
/*                                                nnspp_forward */
/* ---------------------------------------------------------------- */

#define DISPATCH(deviceType, op, type) \
  status = vl::impl::op<deviceType, type>::forward \
  ((type*)output.getMemory(), (type const*)data.getMemory(), \
  data.getHeight(), data.getWidth(), data.getDepth(), data.getSize(), \
  numTotBins,\
  levels.getNumElements(), (type const*)levels.getMemory(),\
  ROIs.getWidth(), (type const *)ROIs.getMemory()) ;


#define DISPATCH2(deviceType, op) \
switch (dataType) { \
case VLDT_Float : DISPATCH(deviceType, op, float) ; break ; \
IF_DOUBLE(case VLDT_Double : DISPATCH(deviceType, op, double) ; break ;) \
default: assert(false) ; return VLE_Unknown ; \
}

#define DISPATCH3(deviceType) \
  switch (method) { \
  case vlSPPAverage : DISPATCH2(deviceType, spp_average) ; break ; \
  case vlSPPMax : DISPATCH2(deviceType, spp_max) ; break ; \
  default: assert(false) ; return VLE_Unknown ; \
  }

ErrorCode
vl::nnspp_forward(vl::Context& context,
                  vl::Tensor output,
                  vl::Tensor data,
                  size_t method,
                  size_t numTotBins,
                  vl::Tensor levels,
                  vl::Tensor ROIs)
{
  ErrorCode status = VLE_Success ;
  vl::DeviceType deviceType = output.getDeviceType();
  vl::DataType dataType = output.getDataType() ;
  switch (deviceType) {
    default:
      assert(false) ;
      return vl::VLE_Unknown ;

  case vl::VLDT_CPU:
	  DISPATCH3(vl::VLDT_CPU);
      break ;

#ifdef ENABLE_GPU
    case vl::VLDT_GPU:
		DISPATCH3(VLDT_GPU) ;
      if (status == VLE_Cuda) {
        context.setError(context.getCudaHelper().catchCudaError(__func__)) ;
      }
      break ;
#endif
  }
  return context.passError(status, "nnspp_forward") ;
}

/* ---------------------------------------------------------------- */
/*                                                   nnspp_backward */
/* ---------------------------------------------------------------- */

#undef DISPATCH
#undef DISPATCH2

// backward max and average want slightly differet argument lists

#define DISPATCH_spp_average(deviceType, type) \
  status = vl::impl::spp_average<deviceType, type>::backward \
  ((type*)derData.getMemory(), (type const*)data.getMemory(), (type const*)derPooled.getMemory(), \
  derData.getHeight(), derData.getWidth(), derData.getDepth(), derData.getSize(), \
  numTotBins, \
  levels.getNumElements(), (const type *)levels.getMemory(), \
  ROIs.getWidth(), (const type *)ROIs.getMemory()) ; \

#define DISPATCH_spp_max(deviceType, type) \
  status = vl::impl::spp_max<deviceType, type>::backward \
  ((type*)derData.getMemory(), (type const*)data.getMemory(), (type const*)derPooled.getMemory(), \
  derData.getHeight(), derData.getWidth(), derData.getDepth(), derData.getSize(), \
  numTotBins, \
  levels.getNumElements(), (const type *)levels.getMemory(), \
  ROIs.getWidth(), (const type *)ROIs.getMemory());

#define DISPATCH2(deviceType, op) \
  switch (dataType) { \
  case VLDT_Float : DISPATCH_ ## op (deviceType, float) ; break ; \
  IF_DOUBLE(case VLDT_Double : DISPATCH_ ## op (deviceType, double) ; break ;) \
  default: assert(false) ; return VLE_Unknown ; \
  }

ErrorCode
vl::nnspp_backward(Context& context,
                   Tensor derData,
                   Tensor data,
                   Tensor derPooled,
                   size_t method,
                   size_t numTotBins,
                   Tensor levels,
                   Tensor ROIs)
{
  vl::ErrorCode status = VLE_Success;
  vl::DeviceType deviceType = derPooled.getDeviceType() ;
  vl::DataType dataType = derPooled.getDataType() ;

  switch (deviceType) {
    default:
      assert(false) ;
	  return vl::VLE_Unknown;

    case vl::VLDT_CPU:
      DISPATCH3(vl::VLDT_CPU) ;
      break ;

#if ENABLE_GPU
	case vl::VLDT_GPU:
		DISPATCH3(vl::VLDT_GPU) ;
      if (status == VLE_Cuda) {
        context.setError(context.getCudaHelper().catchCudaError("spp_*::backward")) ;
      }
      break ;
#endif
  }

  return context.passError(status, "nnspp_backward: ") ;
}
