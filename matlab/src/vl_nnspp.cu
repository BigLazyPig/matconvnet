#include "hip/hip_runtime.h"
// @file spp_cpu.cpp
// @brief SPP block implementation (GPU)
// @author Hakan Bilen 

#include "bits/mexutils.h"
#include "bits/datamex.hpp"
#include "bits/nnspp.hpp"

#if ENABLE_GPU
#include "bits/datacu.hpp"
#endif

#include <assert.h>

/* option codes */
enum {
  opt_numbins=0,
  opt_method,
  opt_verbose,
} ;

/* options */
VLMXOption  options [] = {
  {"NumBins",          1,   opt_numbins },
  {"Method",           1,   opt_method },
  {"Verbose",          0,   opt_verbose },
  {0,                  0,   0           }
} ;

/* ---------------------------------------------------------------- */
/*                                                          Context */
/* ---------------------------------------------------------------- */

vl::MexContext context ;

/*
 Resetting the context here resolves a crash when MATLAB quits and
 the ~Context function is implicitly called on unloading the MEX file.
 */
void atExit()
{
  context.clear() ;
}

/* ---------------------------------------------------------------- */
/*                                                       MEX driver */
/* ---------------------------------------------------------------- */

enum {
  IN_DATA = 0, IN_LEVELS, IN_ROIS, IN_DEROUTPUT, IN_END
} ;

enum {
  OUT_RESULT = 0, OUT_END
} ;

void mexFunction(int nout, mxArray *out[],
                 int nin, mxArray const *in[])
{
  size_t numLevels  = 0;
  size_t numROIs    = 0;
  size_t numTotBins = 0;
  vl::SPPMethod method = vl::vlSPPMax ;

  bool backMode = false ;

  int verbosity = 0 ;
  int opt ;
  int next = IN_END ;
  mxArray const *optarg ;

  /* -------------------------------------------------------------- */
  /*                                            Check the arguments */
  /* -------------------------------------------------------------- */

  mexAtExit(atExit) ;

  if (nin < 3) {
    mexErrMsgTxt("The arguments are less than three.") ;
  }

  if (nin > 3 && vlmxIsString(in[3],-1)) {
    next = 3 ;
    backMode = 0 ;
  } else {
    backMode = (nin >= 4) ;
  }

  while ((opt = vlmxNextOption (in, nin, options, &next, &optarg)) >= 0) {
    switch (opt) {
      case opt_verbose :
        ++ verbosity ;
        break ;
      case opt_numbins :
        if (!vlmxIsPlainMatrix(optarg,-1,-1)) {
          mexErrMsgTxt("NUMBINS is not a plain matrix.") ;
        }
        numTotBins = (size_t)mxGetPr(optarg)[0] ;
        break ;
      case opt_method :
        if (!vlmxIsString(optarg,-1)) {
          vlmxError(VLMXE_IllegalArgument, "METHOD is not a string.") ;
        }
        if (vlmxIsEqualToStringI(optarg, "max")) {
          method = vl::vlSPPMax ;
        } else if (vlmxIsEqualToStringI(optarg, "avg")) {
          method = vl::vlSPPAverage ;
        } else {
          vlmxError(VLMXE_IllegalArgument, "METHOD is not a supported method.") ;
        }
      default:
        break ;
    }
  }


  vl::MexTensor data(context) ;
  vl::MexTensor derOutput(context) ;

  vl::MexTensor rois(context) ;
  vl::MexTensor levels(context) ;

  // load pyramid levels and rois
  levels.init(in[IN_LEVELS]);
  rois.init(in[IN_ROIS]);


  data.init(in[IN_DATA]) ;
  if (backMode) { derOutput.init(in[IN_DEROUTPUT]) ; }

  if (backMode && ! vl::areCompatible(data, derOutput)) {
    mexErrMsgTxt("DATA and DEROUTPUT are not both CPU or GPU arrays.") ;
  }

  numLevels = levels.getNumElements();
  if (numLevels<=0) {
    mexErrMsgTxt("LEVELS has zero elements.") ;
  }

  numROIs = rois.getWidth();

  if ((rois.getHeight() % 5 != 0) || (rois.getNumElements()<=0)) {
    mexErrMsgTxt("ROIs must be a 5xK dimensional array!") ;
  }

  if(numTotBins<=0) {
    mexPrintf("numTotBins %d\n",numTotBins);
    mexErrMsgTxt("numTotBins is wrong.") ;
  }

  if (verbosity > 0) {
    mexPrintf("numTotBins %d depth %d numROIs %d\n",numTotBins,data.getDepth(),numROIs);
  }
  /* Get the output geometry */
  vl::TensorShape outputShape(1, numTotBins,
                              data.getDepth(),
                              numROIs) ;

  vl::TensorShape dataShape = data.getShape();
  dataShape.reshape(4);


  /* Create output buffers */
  vl::DeviceType deviceType = data.getDeviceType() ;
  vl::DataType dataType = data.getDataType() ;
  vl::MexTensor output(context) ;
  vl::MexTensor derData(context) ;


  if (!backMode) {
    output.initWithZeros(deviceType, dataType, outputShape) ;
  } else {
    derData.initWithZeros(deviceType, dataType, dataShape) ;
  }


  if (verbosity > 0) {
    vl::print("vl_nnspp: data: ", data) ;
    if (backMode) {
      vl::print("vl_nnspp: derOutput: ", derOutput) ;
      vl::print("vl_nnspp: derData: ", derData) ;
    } else {
      vl::print("vl_nnspp: output: ", output) ;
      vl::print("vl_nnspp: rois: ", rois) ;
      vl::print("vl_nnspp: levels: ", levels) ;
    }
  }




  if (verbosity > 0) {
    mexPrintf("vl_spp: %s; %s", backMode?"backward":"forward", (data.getDeviceType()==vl::VLDT_GPU) ? "GPU" : "CPU") ;
    mexPrintf("\nvl_spp: method %d numLevels %d; numROIs %d numTotBins %d\n", method, numLevels, numROIs, numTotBins);
  }

  /* -------------------------------------------------------------- */
  /*                                                    Do the work */
  /* -------------------------------------------------------------- */

  vl::ErrorCode error ;
  if (!backMode) {
    error = vl::nnspp_forward(context,
                              output, data,
                              method,
                              numTotBins,
                              levels,
                              rois) ;

  } else {
    error = vl::nnspp_backward(context,
                               derData, data, derOutput,
                               method,
                               numTotBins,
                               levels,
                               rois) ;
  }

  /* -------------------------------------------------------------- */
  /*                                                         Finish */
  /* -------------------------------------------------------------- */

  if (error != vl::VLE_Success) {
    mexErrMsgTxt(context.getLastErrorMessage().c_str()) ;
  }
  if (backMode) {
    out[OUT_RESULT] = derData.relinquish() ;
  } else {
    out[OUT_RESULT] = output.relinquish() ;
  }
}
